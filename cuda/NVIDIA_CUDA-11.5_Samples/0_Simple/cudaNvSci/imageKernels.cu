#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2019 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return ((unsigned int)(rgba.w * 255.0f) << 24) |
         ((unsigned int)(rgba.z * 255.0f) << 16) |
         ((unsigned int)(rgba.y * 255.0f) << 8) |
         ((unsigned int)(rgba.x * 255.0f));
}

////////////////////////////////////////////////////////////////////////////////
//! Rotate an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
static __global__ void transformKernel(unsigned int *outputData, int width,
                                       int height, float theta,
                                       hipTextureObject_t tex) {
  // calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = (float)x - (float)width / 2;
  float v = (float)y - (float)height / 2;
  float tu = u * cosf(theta) - v * sinf(theta);
  float tv = v * cosf(theta) + u * sinf(theta);

  tu /= (float)width;
  tv /= (float)height;

  // read from texture and write to global memory
  float4 pix = tex2D<float4>(tex, tu + 0.5f, tv + 0.5f);
  unsigned int pixelInt = rgbaFloatToInt(pix);
  outputData[y * width + x] = pixelInt;
}

static __global__ void rgbToGrayscaleKernel(unsigned int *rgbaImage,
                                            size_t imageWidth,
                                            size_t imageHeight) {
  size_t gidX = blockDim.x * blockIdx.x + threadIdx.x;

  uchar4 *pixArray = (uchar4 *)rgbaImage;

  for (int pixId = gidX; pixId < imageWidth * imageHeight;
       pixId += gridDim.x * blockDim.x) {
    uchar4 dataA = pixArray[pixId];
    unsigned char grayscale =
        (unsigned char)(dataA.x * 0.3 + dataA.y * 0.59 + dataA.z * 0.11);
    uchar4 dataB = make_uchar4(grayscale, grayscale, grayscale, 0);
    pixArray[pixId] = dataB;
  }
}

void launchGrayScaleKernel(unsigned int *d_rgbaImage,
                           std::string image_filename, size_t imageWidth,
                           size_t imageHeight, hipStream_t stream) {
  int numThreadsPerBlock = 1024;
  int numOfBlocks = (imageWidth * imageHeight) / numThreadsPerBlock;

  rgbToGrayscaleKernel<<<numOfBlocks, numThreadsPerBlock, 0, stream>>>(
      d_rgbaImage, imageWidth, imageHeight);

  unsigned int *outputData;
  checkCudaErrors(hipHostMalloc((void **)&outputData, sizeof(unsigned int) * imageWidth * imageHeight));
  checkCudaErrors(hipMemcpyAsync(
      outputData, d_rgbaImage, sizeof(unsigned int) * imageWidth * imageHeight,
      hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  char outputFilename[1024];
  strcpy(outputFilename, image_filename.c_str());
  strcpy(outputFilename + image_filename.length() - 4, "_out.ppm");
  sdkSavePPM4ub(outputFilename, (unsigned char *)outputData, imageWidth,
                imageHeight);
  printf("Wrote '%s'\n", outputFilename);

  checkCudaErrors(hipHostFree(outputData));
}

void rotateKernel(hipTextureObject_t &texObj, const float angle,
                  unsigned int *d_outputData, const int imageWidth,
                  const int imageHeight, hipStream_t stream) {
  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(imageWidth / dimBlock.x, imageHeight / dimBlock.y, 1);

  transformKernel<<<dimGrid, dimBlock, 0, stream>>>(d_outputData, imageWidth,
                                                    imageHeight, angle, texObj);
}
